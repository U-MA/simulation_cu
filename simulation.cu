#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include <iostream>
using namespace std;


struct route
{
    int routeSize;
    int *route;
    int vehicleNumber;
    int *visitCustomer; // 訪れた顧客の数
    int *quantity;
};

typedef struct route * Route;


__global__ void randomSimulation(Route rdata, int *rewards)
{
    const int customerSize = Vrp_GetNumberOfCustomers();
    int candidates[customerSize], candidateSize;

    // thread数がcustomerより多いことを想定
    int customer = threadIdx.x;

    while (!Route_AllCustomersIsVisited(rdata) && Vrp_VehicleIsInBound())
    {
        // 訪問していない顧客を調べる
        if (customer < customerSize)
        {
            if (isVisit(rdata, customer))
            {
                // candidates配列にシーケンシャルに代入する
                // candidateSizeをインクリメントする
            }
        }

        __syncthreads();

        // 一つのthreadだけがすればよい
        if (threadIdx.x == 0)
        {
            if (candidateSize != 0)
            {
                // rand()関数をMTGP(?)にする必要がある
                int elected = rand() % candidateSize;
                Route_Update(rdata, rdata->vehicleNumber);
            }
            else
            {
                Route_SetVehicleNumber(rdata, rdata->vehicleNumber+1);
            }
        }
    }

    if (threadIdx.x == 0)
    {
        if (Route_AllCustomersIsVisited(rdata))
            rewards =  Route_CalculateCost(rdata);
        else
            rewards = 100000;
    }
}


// 要素数nの配列aの中から最小値を求める
__global__ reduction(int *a, int n, int *b)
{
}


int main(int argc, char **argv)
{
    Vrp_Create("Vrp-All/E/E-n13-k4.vrp");
    Route rdata = Route_Create();
    Route dev_rdata;
    int reward, *dev_reward, *dev_rewards;

    const int blocks = 1024;
    const int threads = Vrp_GetNumberOfCustomers();

    hipMalloc((void**)&dev_rdata, sizeof(struct route));
    hipMalloc((void**)&dev_rewards, sizeof(int) * blocks);
    hipMalloc((void**)&dev_reward, sizeof(int));

    hipMemcpy(dev_rdata, rdata, sizeof(struct route), hipMemcpyHostToDevice);

    randomSimulation<<<1024,threads>>>(dev_route, dev_rewards);
    reduction<<<1,blocks>>>(dev_rewards, blocks, dev_reward);

    hipMemcpy(reward, dev_reward, sizeof(int), hipMemcpyDeviceToHost);

    cout << "reward: " << reward << endl;

    return 0;
}
