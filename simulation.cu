#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include <iostream>
using namespace std;


struct route
{
    int routeSize;
    int *route;
    int vehicleNumber;
    int *visitCustomer; // 訪れた顧客の数
    int *quantity;
	__device__ bool isVisit(const int customer)
	{
		for (int i=0; i < route_size; i++)
		{
			if (route[i] == customer) return true;
		}
		return false;
	}
	__device__ bool isVisitedAll(const int customer_size)
	{
		for (int i=1; i < customer_size; i++)
		{
			for (int j=0; j < routeSize; j++)
			{
				if (route[j] == i) break;
			}
			if (j == routeSize) return false;
		}
		return true;
	}
	__device__ void update(int move)
	{
		if (move == 0)
		{
			vehicleNumber++;
			return;
		}

		int offset = // TODO;
		route[offset] = move;
		VisitedCustomer[vehicleNumber]++;
		quantity[vehicleNumber] += //TODO: capacityの追加
	}
	__device__ void vehicleChange()
	{
		vehicleNumber++;
	}
	__device__ unsigned int calculateCost()
	{
		return 0; // TODO
	}
};

typedef struct route * Route;


/* TODO: Vrpモジュールをどう扱うか
 *       2013.11.13現在 Vrpモジュールはシングルインスタンスモジュールとして
 *                      vrp.cpp内にファイルスコープに入れている
 *                      GPUコードでは扱いにくいので公開するべきか？
 *                      それともvrp.cppをvrp.cuに変更し、すべての関数に
 *                      __device__ __host__をつけるべきか？
 */
__global__ void randomSimulation(Route rdata, unsigned int *rewards)
{
    const int customerSize = Vrp_GetNumberOfCustomers();
    int candidates[customerSize], candidateSize;

    // thread数がcustomerより多いことを想定
    int customer = threadIdx.x;

    while (!rdata->isVisitedAll(customerSize) && Vrp_VehicleIsInBound())
    {
        // 訪問していない顧客を調べる
        if (customer < customerSize)
        {
            if (rdata->isVisit(customer))
            {
                // candidates配列にシーケンシャルに代入する
                // candidateSizeをインクリメントする
            }
        }

        __syncthreads();

        // rdataに顧客を追加するor車体の変更
		//一つのthreadだけがすればよい
        if (threadIdx.x == 0)
        {
            if (candidateSize != 0)
            {
                // rand()関数をrandom123にする必要がある
                int elected = rand() % candidateSize;
                rdata->update(elected);
            }
            else
            {
                rdata->vehicleChange();
            }
        }
    }

    if (threadIdx.x == 0)
    {
        if (rdata->isVisitedAll(customerSize))
            *rewards =  Route_CalculateCost(rdata);
        else
            *rewards = 100000;
    }
}


// 要素数nの配列aの中から最小値を求める
__global__ reduction(int *a, int n, int *b)
{
}


int main(int argc, char **argv)
{
    Vrp_Create("Vrp-All/E/E-n13-k4.vrp");
    Route rdata = Route_Create();
    Route dev_rdata;
    int reward, *dev_reward, *dev_rewards;

    const int blocks = 1024;
    const int threads = Vrp_GetNumberOfCustomers();

    hipMalloc((void**)&dev_rdata, sizeof(struct route));
    hipMalloc((void**)&dev_rewards, sizeof(int) * blocks);
    hipMalloc((void**)&dev_reward, sizeof(int));

    hipMemcpy(dev_rdata, rdata, sizeof(struct route), hipMemcpyHostToDevice);

    randomSimulation<<<1024,threads>>>(dev_route, dev_rewards);
    reduction<<<1,blocks>>>(dev_rewards, blocks, dev_reward);

    hipMemcpy(reward, dev_reward, sizeof(int), hipMemcpyDeviceToHost);

    cout << "reward: " << reward << endl;

    return 0;
}
